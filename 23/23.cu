#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define MAX_STEPS 100000

// On bourrine monte carto sur le GPU.
// Ca marche pas mais c'est fun.
__global__ void monteCarloMazeKernel(char* maze, int width, int height, int* results) {
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    bool visited[MAX_STEPS] = { false };
    bool found = false;
    int pathLength = 0;
    int x = 1; // Point de départ en x (colonne)
    int y = 0; // Point de départ en y (ligne)
    hiprandState state;
    hiprand_init(1234 + threadId, 0, 0, &state);

    visited[y * width + x] = true; // Marquer la position de départ comme visitée

    for (int steps = 0; steps < MAX_STEPS; steps++) {
        int validDirections[4] = { -1, -1, -1, -1 };
        int numValidDirections = 0;

        for (int direction = 0; direction < 4; direction++) {
            int dx = 0, dy = 0;
            switch (direction) {
            case 0: dy = -1; break; // Nord
            case 1: dx = 1; break; // Est
            case 2: dy = 1; break; // Sud
            case 3: dx = -1; break; // Ouest
            }

            int newX = x + dx;
            int newY = y + dy;

            if (newX >= 0 && newX < width && newY >= 0 && newY < height &&
                maze[newY * width + newX] != '#' && !visited[newY * width + newX]) {
                validDirections[numValidDirections++] = direction;
            }
        }

        if (numValidDirections == 0) {
            results[threadId] = -1;
            return;
        }

        int chosenDirection = validDirections[int(hiprand_uniform(&state) * numValidDirections)];
        switch (chosenDirection) {
        case 0: y--; break;
        case 1: x++; break;
        case 2: y++; break;
        case 3: x--; break;
        }

        if (x >= 0 && x < width && y >= 0 && y < height && maze[y * width + x] != '#') {
            visited[y * width + x] = true;
            pathLength++;
        }

        if (x == width - 2 && y == height - 1) {
            found = true;
            break;
        }
    }

    if (found) {
        printf("%d: %d\n", threadId, pathLength);
        results[threadId] = pathLength;
    }
    else {
        results[threadId] = -1;
    }
}

std::vector<std::string> readMaze(const std::string& filename) {
    std::vector<std::string> maze;
    std::ifstream file(filename);
    std::string line;

    while (std::getline(file, line)) {
        maze.push_back(line);
    }

    return maze;
}

std::vector<char> linearizeMaze(const std::vector<std::string>& maze) {
    std::vector<char> linearMaze;
    for (const std::string& line : maze) {
        for (char c : line) {
            linearMaze.push_back(c);
        }
    }
    return linearMaze;
}

int main(int argc, char* argv[]) {
    std::vector<std::string> maze = readMaze("input");
    std::vector<char> linearMaze = linearizeMaze(maze);

    int width = maze[0].size();
    int height = maze.size();

    char* dev_maze;
    size_t size = linearMaze.size() * sizeof(char);
    hipMalloc((void**)&dev_maze, size);
    hipMemcpy(dev_maze, linearMaze.data(), size, hipMemcpyHostToDevice);

    const int numThreads = 65535; // lol
    dim3 blockSize(512);
    dim3 gridSize((numThreads + blockSize.x - 1) / blockSize.x);

    int* dev_results;
    hipMalloc((void**)&dev_results, numThreads * sizeof(int));

    monteCarloMazeKernel<<<gridSize, blockSize>>>(dev_maze, width, height, dev_results);

    int* host_results = new int[numThreads];
    hipMemcpy(host_results, dev_results, numThreads * sizeof(int), hipMemcpyDeviceToHost);

    int maxLength = 0;
    for (int i = 0; i < numThreads; i++) {
        if (host_results[i] > maxLength) {
            maxLength = host_results[i];
        }
    }
    std::cout << "Part 2: " << maxLength << std::endl;

    delete[] host_results;
    hipFree(dev_results);
    hipFree(dev_maze);

    return 0;
}
